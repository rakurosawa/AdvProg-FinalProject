
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define SIZE_OF_ARRAY 16
#define NUM_THDS 8
#define BLOCK_SIZE 1


__global__ void gpuSelectionSort(int* array){
    __shared__ int temp[SIZE_OF_ARRAY];

    int globalIdx = threadIdx.x + (BLOCK_SIZE * blockIdx.x);


    for (int i = 0; i < SIZE_OF_ARRAY; i ++){
        __syncthreads();

        int minIdx = i;

        //printf("thread: %d\n", globalIdx);

        int arrIdx1 = 2 * globalIdx + i;
        int arrIdx2 = arrIdx1 + 1;
        //printf("arrIdx1: %d\n", arrIdx1);

        if (arrIdx1 >= SIZE_OF_ARRAY){
            temp[globalIdx * 2] = array[SIZE_OF_ARRAY - 1];
        }
        else {
            temp[globalIdx * 2] = array[arrIdx1];
        }
        
        if (arrIdx2 >= SIZE_OF_ARRAY){
            temp[globalIdx * 2 + 1] = array[SIZE_OF_ARRAY - 1];
        }
        else {
            temp[globalIdx * 2 + 1] = array[arrIdx2];
        }

        for (int j = NUM_THDS; j > 0; j /= 2){
            __syncthreads();
            if (globalIdx <= j){
                int idx1 = globalIdx * 2;
                int val1 = temp[idx1];
                int idx2 = idx1 + 1;
                int val2 = temp[idx2];

                if (val1 < val2){
                    temp[globalIdx] = val1;
                }
                else {
                    temp[globalIdx] = val2;
                }
            }   
        }

        for (int k = 0; k < SIZE_OF_ARRAY; k ++){
            if (array[k] == temp[0]){
                minIdx = k;
            }
        }

        if (temp[0] <= array[i]){
            int tempVal = array[i];
            array[i] = array[minIdx];
            array[minIdx] = tempVal;
        }
    }
    __syncthreads();
}
    


int main(){

    int *input;
    hipMallocManaged(&input, (SIZE_OF_ARRAY)*sizeof(int));

    for (int i = 0; i < SIZE_OF_ARRAY; i ++){
        input[i] = SIZE_OF_ARRAY - i;
    }

    for (int i = 0; i < SIZE_OF_ARRAY; i ++){
        printf("%d  ", input[i]);
    }
    printf("\n");


    // <<< number of blocks, size of each block >>>
    gpuSelectionSort<<<BLOCK_SIZE, NUM_THDS>>>(input);
    hipDeviceSynchronize();

    for (int i = 0; i < SIZE_OF_ARRAY; i ++){
        printf("%d  ", input[i]);
    }
    printf("\n");

    // cudaFree(histogram);
}